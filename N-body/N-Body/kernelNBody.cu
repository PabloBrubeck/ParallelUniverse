#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <cutil_math.h>
#include "geometry.h"
#include "linalg.h"
#include "Planck.h"

#define MAXTHREADS 512


__device__
struct Particle{
	float mass;
	float3 pos;
	float3 vel;
	float3 acc;
};


// device functions
__device__
float mod(float x, float y) {
	return x-y*floor(x/y);
}
__device__
float invsqrt(float x){
	long i;
	float x2, y;
	const float threehalfs = 1.5F;
	x2=x*0.5F;
	y=x;
	i=*(long*)&y;                // evil floating point bit level hacking
	i=0x5f3759df-(i>>1);         // what the fuck?
	y=*(float*)&i;
	y=y*(threehalfs-(x2*y*y));   // 1st iteration
    y=y*(threehalfs-(x2*y*y));   // 2nd iteration, this can be removed
	return y;
}
__device__
void bodyBodyInteraction(float3 &acc, float3 &p, float4 &q){
	float3 r=make_float3(q.x-p.x, q.y-p.y, q.z-p.z);
	float r2=dot(r, r);
	float w2=q.w*invsqrt(r2*r2*r2+0.0006f);
	acc+=r*w2;
}


// simulation kernels
__global__
void initPos(uchar4 *d_color, Particle *d_body, dim3 mesh){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<mesh.x && j<mesh.y){
		int k=j*mesh.x+i;

		float theta=(2*i*PI)/mesh.x;
		float h=float(j+1)/mesh.y;
		
		float cos=cosf(theta);
		float sin=sinf(theta);

		float a=1.0f*h, b=1.5f*h;
		float x=a*cos;
		float y=b*sin;

		float r=sqrtf(x*x+y*y);
		float m=12*(1+3*r)*exp(-3*r);

		d_body[k].mass=m;
		d_body[k].pos={x, y, 0.f};
		
		float temp=1000.f*m;
		d_color[k]=planckColor(temp);
	}
}
__global__
void initVel(Particle *d_body, dim3 mesh){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<mesh.x && j<mesh.y){
		int k=j*mesh.x+i;
		
		float theta=(2*i*PI)/mesh.x;
		float h=float(j+1)/mesh.y;

		float a=1.0f*h, b=1.5f*h;
		float3 g=d_body[k].acc;
		float3 r=d_body[k].pos;

		float3 p=make_float3(r.x/(a*a), r.y/(b*b), 0.f);
		float w2=-dot(g, p);
		float w=sqrtf(max(0.f, w2));

		d_body[k].vel={-a/b*w*r.y, b/a*w*r.x, 0.f};
	}
}
__global__
void interact(Particle *d_body, int n){
	extern __shared__ float4 s_buff[];
	int tid=threadIdx.x;
	int src=blockIdx.x*blockDim.x+tid;
	int dst=blockIdx.y*blockDim.x+tid;
	if(src<n){
		s_buff[tid]=make_float4(d_body[src].pos, d_body[src].mass);
	}
	__syncthreads();
	if(dst<n){
		float3 pos=d_body[dst].pos;
		float3 acc=make_float3(0.f, 0.f, 0.f);
		for(int i=0; i<blockDim.x; i++){
			bodyBodyInteraction(acc, pos, s_buff[i]);
		}
		atomicAdd(&(d_body[dst].acc.x), acc.x);
		atomicAdd(&(d_body[dst].acc.y), acc.y);
		atomicAdd(&(d_body[dst].acc.z), acc.z);
	}
}
__global__
void integrate(Particle *d_body, float dt, int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float3 vel=d_body[i].vel+d_body[i].acc*dt;
		d_body[i].acc=make_float3(0.f, 0.f, 0.f);
		d_body[i].vel=vel;
		d_body[i].pos+=vel*dt;
	}
}
__global__
void updatePoints(float4 *d_pos, float4 *d_norm, Particle *d_body, int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		d_pos[i]=make_float4(d_body[i].pos, 1.f);
		d_norm[i]=make_float4(normalize(d_body[i].vel), 1.f);
	}
}


// auxiliary kernels
__global__
void mapMagnitude2(Particle *d_body, float *d_abs, int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float3 acc=d_body[i].acc;
		d_abs[i]=dot(acc, acc);
	}
}
__global__
void reduceMax(float *d_in, float *d_out, int n){   
    extern __shared__ float shared[];
	int tid=threadIdx.x;
    int gid=blockIdx.x*blockDim.x+tid;
	shared[tid]= gid<n? d_in[gid]: -FLT_MAX;
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s){
            shared[tid]=__max(shared[tid], shared[tid+s]);
        }
        __syncthreads();
    }
    if(tid==0){
        d_out[blockIdx.x]=shared[0];
    }
}


// host functions
int ceil(int num, int den){
	return (num+den-1)/den;
}
unsigned int nextPowerOf2(unsigned int n){
  unsigned k=0;
  if(n&&!(n&(n-1))){
	  return n;
  }
  while(n!=0){
    n>>=1;
    k++;
  }
  return 1<<k;
}
float getMax(float *d_in, const size_t numElems){
	int n=numElems;
	int grid, block=MAXTHREADS;
	float *h_out=new float();
	do{
		grid=(n+block-1)/block;
		if(grid==1){
			block=nextPowerOf2(n);
		}
		reduceMax<<<grid, block, block*sizeof(float)>>>(d_in, d_in, n);
		n=grid;
	}while(grid>1);
	checkCudaErrors(hipMemcpy(h_out, d_in, sizeof(float), hipMemcpyDeviceToHost));
	return *h_out;
}


// kernel invocator
void launch_kernel(float4 *d_pos, float4 *d_norm, uchar4 *d_color, uint4 *d_index, dim3 mesh, float time){
	static const int n=mesh.x*mesh.y*mesh.z;
	static const int block1D=MAXTHREADS;
	static const int grid1D=ceil(n, block1D);
	static const int p=128;
	static const int bytes=p*sizeof(float4);
	static const dim3 grid2D(ceil(n, p), ceil(n, p));

	static const float dvmax=1.f;
	static Particle *d_body=NULL;
	static float *d_aux=NULL;

	if(d_body==NULL){
		// initialization
		checkCudaErrors(hipMalloc((void**)&d_aux, n*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_body, n*sizeof(Particle)));
		dim3 tblock(16, 16);
		dim3 tgrid(ceil(mesh.x, tblock.x), ceil(mesh.y, tblock.y));
		
		initPos<<<tgrid, tblock>>>(d_color, d_body, mesh);
		interact<<<grid2D, p, bytes>>>(d_body, n);
		initVel<<<tgrid, tblock>>>(d_body, mesh);
	}

	// main kernel sequence
	interact<<<grid2D, p, bytes>>>(d_body, n);
	mapMagnitude2<<<grid1D, block1D>>>(d_body, d_aux, n);
	float dt=dvmax/sqrt(getMax(d_aux, n));
	integrate<<<grid1D, block1D>>>(d_body, dt, n);
	updatePoints<<<grid1D, block1D>>>(d_pos, d_norm, d_body, n);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}