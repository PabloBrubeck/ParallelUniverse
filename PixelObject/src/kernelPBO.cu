#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuColor.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>

#define MAXTHREADS 512
#define COLORDEPTH 256

uchar4 *d_cmap;
float *d_u, *d_ul, *d_lap;
double2 axes=make_double2(4.0f, 4.0f);
double2 origin=make_double2(-2.0f, -2.0f);

inline int ceil(int num, int den){
	return (num+den-1)/den;
}

__global__ void laplacian(float *d_lap, float *d_u, int2 image){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int ip=(i+1)%image.x;
		int im=(i-1+image.x)%image.x;
		int jp=(j+1)%image.y;
		int jm=(j-1+image.y)%image.y;
		d_lap[i*image.y+j]=d_u[ip*image.y+j]
				+d_u[im*image.y+j]
				+d_u[i*image.y+jp]
				+d_u[i*image.y+jm]
				-4*d_u[i*image.y+j];
	}
}
__global__ void heatSolve(uchar4 *d_pixel, uchar4 *d_cmap, float *d_u, float *d_lap, float c, double2 origin, double2 axes, int2 image, float time){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		d_u[gid]+=c*d_lap[gid];
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);
		if(x*x+y*y<1.E-2f){
			d_u[gid]=1.f;
		}
		if(i==0 || j==0 || i==image.x-1 || j==image.y-1){
			d_u[gid]=0.f;
		}
		int k=(int)(COLORDEPTH*d_u[gid]);
		d_pixel[gid]=d_cmap[clamp(k, 0, COLORDEPTH-1)];
	}
}
void heatPDE(uchar4 *d_pixel, uchar4 *d_cmap, float *d_u, float *d_lap, double2 origin, double2 axes, int2 image, float time){
	static dim3 block(MAXTHREADS);
	static dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	laplacian<<<grid, block>>>(d_lap, d_u, image);
	heatSolve<<<grid, block>>>(d_pixel, d_cmap, d_u, d_lap, 0.2f, origin, axes, image, time);
}

__global__ void waveSolve(uchar4 *d_pixel, uchar4 *d_cmap, float *d_u, float *d_ul, float *d_lap, float c, double2 origin, double2 axes, int2 image, float time){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		float temp=d_u[gid];
		d_u[gid]=2*temp-d_ul[gid]+c*d_lap[gid];
		d_ul[gid]=temp;
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);
		if(max(abs(x),abs(y))<1.E-2f){
			d_u[gid]=sinpif(time);
		}
		if(i==0 || j==0 || i==image.x-1 || j==image.y-1){
			d_u[gid]=0.f;
		}
		int k=(int)(COLORDEPTH*(1+d_u[gid])/2);
		d_pixel[gid]=d_cmap[clamp(k, 0, COLORDEPTH-1)];
	}
}
void wavePDE(uchar4 *d_pixel, uchar4 *d_cmap, float *d_u, float *d_ul, float *d_lap, double2 origin, double2 axes, int2 image, float time){
	static dim3 block(1,MAXTHREADS);
	static dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	laplacian<<<grid, block>>>(d_lap, d_u, image);
	waveSolve<<<grid, block>>>(d_pixel, d_cmap, d_u, d_ul, d_lap, 0.4f, origin, axes, image, time);
}


__device__ int interference(float x, float y, float t){
	float r1=sqrtf((x-1)*(x-1)+y*y);
	float r2=sqrtf((x+1)*(x+1)+y*y);
	float z1=expf(-r1*r1/4)*sinpif(5*r1-t);
	float z2=expf(-r2*r2/4)*sinpif(5*r2-t);
	return (int)(COLORDEPTH*(1.4351+z1+z2)/2.98489);
}
__device__ int mandelbrotf(float x, float y){
	hipComplex c=make_hipComplex(x, y);
	hipComplex z=make_hipComplex(x, y);
	int k=0;
	float z2=0.0;
	while(z2<4.f && k<COLORDEPTH){
		z=hipCaddf(hipCmulf(z, z), c);
		z2=z.x*z.x+z.y*z.y;
		k++;
	}
	return k-1;
}
__device__ int mandelbrotd(double x, double y){
	hipDoubleComplex c=make_hipDoubleComplex(x, y);
	hipDoubleComplex z=make_hipDoubleComplex(x, y);
	int k=0;
	double z2=0.0;
	while(z2<4.f && k<COLORDEPTH){
		z=hipCadd(hipCmul(z, z), c);
		z2=z.x*z.x+z.y*z.y;
		k++;
	}
	return k-1;
}

__global__ void kernelf(uchar4* d_pixel, uchar4 *d_cmap, double2 origin, double2 axes, int2 image, float time){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);
		int k=mandelbrotf(x, y);
		d_pixel[gid]=d_cmap[clamp(k,0,COLORDEPTH-1)];
	}
}
__global__ void kerneld(uchar4* d_pixel, uchar4 *d_cmap, double2 origin, double2 axes, int2 image, float time){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int k, gid=j*image.x+i;
		double x=(double)i/image.x*axes.x+origin.x;
		double y=(double)j/image.y*axes.y+origin.y;
		k=mandelbrotd(x, y);
		d_pixel[gid]=d_cmap[clamp(k,0,COLORDEPTH-1)];
	}
}



void init_kernel(int2 image){
	int n=image.x*image.y;
	checkCudaErrors(hipMalloc((void**)&d_u, n*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_ul, n*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_lap, n*sizeof(float)));
	checkCudaErrors(hipMemset(d_u, 0.f, n*sizeof(float)));
	checkCudaErrors(hipMemset(d_ul, 0.f, n*sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&d_cmap, COLORDEPTH*sizeof(uchar4)));
	jet<<<1, COLORDEPTH>>>(d_cmap, COLORDEPTH);
}

void launch_kernel(uchar4* d_pixel, int2 image, float time){
	static const dim3 block(MAXTHREADS);
	static const dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	wavePDE(d_pixel, d_cmap, d_u, d_ul, d_lap, origin, axes, image, time);
	//kerneld<<<grid, block>>>(d_pixel, d_cmap, origin, axes, image, time);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
