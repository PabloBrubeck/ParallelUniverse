#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "cuMatlab.h"
#include "examples.h"
#include "animation.h"

using namespace std;

void imageExample(const int w, const int h){
	uchar4 *d_rgba;
	hipMalloc((void**)&d_rgba, w*h*sizeof(uchar4));

	auto dcolor = [] __device__ (double x, double y){
		hipDoubleComplex z=make_hipDoubleComplex(x, y);
		hipDoubleComplex w=asin(z);
		return hsv2rgb(angle(w)/(2*pi),1,1);
	};

	double L=3*pi;
	double xmin=-L, xmax=L, ymin=-h*L/w, ymax=h*L/w;
	cudaMap(dcolor, w, h, d_rgba, w, xmin, xmax, ymax, ymin);

	string path="/home/pbrubeck/ParallelUniverse/cuMatlab/data/DomainColor.png";
	imwrite(w,h,d_rgba,path);
	hipFree(d_rgba);
}


__global__ void mandelbrotf(int m, int n, uchar4* d_rgba, float x1, float x2, float y1, float y2){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<m && j<n){
		float x=x1+i*(x2-x1)/(m-1);
		float y=y1+j*(y2-y1)/(n-1);
		hipComplex z=make_hipComplex(x,y);
		hipComplex w=make_hipComplex(x,y);
		int k=0;
		while(k<64 && w.x*w.x+w.y*w.y<4){
			w=hipCfmaf(w,w,z);
			k++;
		};
		float h=(k%65)/65.f;
		d_rgba[j*m+i]=jet(h);
	}
}

void fractal(int w, int h, uchar4* d_rgba, double x1, double x2, double y1, double y2){
	mandelbrotf<<<grid(w,h), MAXTHREADS>>>(w, h, d_rgba, x1, x2, y2, y1);
}


int main(int argc, char **argv){
	//animation(argc, argv, 1024, 1024, fractal);
	//auto f=[] __device__ (double x){return sinpi(x);};
	//poisson(f, -1, 1, 32);

	waveExample(512);

	printf("Program terminated.\n");
	return 0;
}
