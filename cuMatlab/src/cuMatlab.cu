#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuMatlab.h"
#include "WaveSolver.h"

using namespace std;

void waveExample(int N){
	double *u=new double[N];
	linspace(u, -pi, pi, N);
	auto f=[](double th)->double{return th>0?0:(1-cos(2*th))/2;};
	map(u, u, N, f);
	double *d_u;
	hipMalloc((void**)&d_u, 2*N*sizeof(double));
	hipMemset(d_u, 0.0, 2*N*sizeof(double));
	hipMemcpy(d_u, u, N*sizeof(double), hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	WaveSolver wave(handle, N, d_u, 0.0);

	int frames=144*10;
	double dt=6.0/(N*N);
	for(int i=0; i<frames; i++){
		wave.solve(dt);
	}
	hipMemcpy(u, d_u, N*sizeof(double), hipMemcpyDeviceToHost);
	disp(u, N, 1, 1);
	hipblasDestroy(handle);
}

void cufftExample(int N){
	double *u=new double[N];
	linspace(u, -pi, pi, N);
	auto f=[](double th)->double{return cos(th);};
	map(u, u, N, f);
	double *d_u;
	hipMalloc((void**)&d_u, N*sizeof(double));
	hipMemcpy(d_u, u, N*sizeof(double), hipMemcpyHostToDevice);

	hipfftHandle fftPlan, ifftPlan;
	hipfftPlan1d(&fftPlan, N, HIPFFT_D2Z, 1);
	hipfftPlan1d(&ifftPlan, N, HIPFFT_Z2D, 1);

	hipfftDoubleComplex *d_uhat;
	hipMalloc((void**)&d_uhat, N*sizeof(hipfftDoubleComplex));

	fftD(fftPlan, ifftPlan, N, 1, d_u, d_u, d_uhat);
	hipMemcpy(u, d_u, N*sizeof(double), hipMemcpyDeviceToHost);
	disp(u, N, 1, 1);

	hipfftDestroy(fftPlan);
	hipfftDestroy(ifftPlan);
}

void mapExample(int N){
	double *x=new double[N];
	double *y=new double[N];
	linspace(x, 0, pi, N);
	int m=0, deg=5;
	double *a=new double[deg]; a[deg-1]=1;
	auto f1=[a,deg,m](double th)->double{return LegendreP(a, deg, m, cos(th));};
	map(y, x, N, f1);
	disp(y,N,1,1);
}

void poisson(double ua, double ub, int n){
	double *d_x, *d_D, *d_D2;
	hipMalloc((void**)&d_x, n*sizeof(double));
	hipMalloc((void**)&d_D, n*n*sizeof(double));
	hipMalloc((void**)&d_D2, n*n*sizeof(double));
	chebD(n, d_D, d_x);

	// compute second derivative operator D2=D*D
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);
	double alpha=1;
	double beta=0;
	hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_D, n, d_D, n, &beta, d_D2, n);

	// right hand side
	// TODO: map on device, heterogeneous lambda
	double *h_b=new double[n];
	auto f=[](double x)->double{return 1.0;};
	map(h_b, h_b, n, f);
	double *d_b;
	hipMalloc((void**)&d_b, n*sizeof(double));
	hipMemcpy(d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);

	alpha=-ua;
	hipblasDaxpy(cublasH, n, &alpha, d_D2+n*(n-1), 1, d_b, 1);
	alpha=-ub;
	hipblasDaxpy(cublasH, n, &alpha, d_D2        , 1, d_b, 1);
	hipMemcpy(d_b,     &ub, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b+n-1, &ua, sizeof(double), hipMemcpyHostToDevice);

	// solve Poisson D2*u=f(x)
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);
	int lwork;
	int m=n-2, lda=n, ldb=n, nrhs=1;
	hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_D2+n+1, lda, &lwork);

	int    *d_info; hipMalloc((void**)&d_info, sizeof(int));
	int    *d_Ipiv; hipMalloc((void**)&d_Ipiv, m*sizeof(int));
	double *d_Work; hipMalloc((void**)&d_Work, lwork*sizeof(double));

	hipsolverDnDgetrf(cusolverH, m, m, d_D2+n+1, lda, d_Work, d_Ipiv, d_info);
	hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, nrhs, d_D2+n+1, lda, d_Ipiv, d_b+1, ldb, d_info);

	// display u
	hipMemcpy(h_b, d_b, n*sizeof(double), hipMemcpyDeviceToHost);
	disp(h_b, n, 1, 1);
	free(h_b);

	// free auxiliary variables
	hipFree(d_Work);
	hipFree(d_Ipiv);
	hipFree(d_info);

	// free device memory
	hipFree(d_x);
	hipFree(d_D);
	hipFree(d_D2);
	hipFree(d_b);

	// destroy library handles
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);
}

int main(int argc, char **argv){
	waveExample(1<<8);
	return 0;
}
