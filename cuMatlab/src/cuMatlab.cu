#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuMatlab.h"

using namespace std;

void mapExample(int N){
	double *x=new double[N];
	double *y=new double[N];
	linspace(x, 0, pi, N);
	int m=0, deg=5;
	double *a=new double[deg]; a[deg-1]=1;
	auto f1=[a,deg,m](double th)->double{return LegendreP(a, deg, m, cos(th));};
	map(y, x, N, f1);
	disp(y,N,1,1);
}

void poisson(double ua, double ub, int n){
	double *d_x, *d_D, *d_D2;
	hipMalloc((void**)&d_x, n*sizeof(double));
	hipMalloc((void**)&d_D, n*n*sizeof(double));
	hipMalloc((void**)&d_D2, n*n*sizeof(double));
	chebD(d_D, d_x, n);

	// compute second derivative operator D2=D*D
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);
	double alpha=1, beta=0;
	hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_D, n, d_D, n, &beta, d_D2, n);

	// right hand side
	double *h_b=new double[n];
	auto f=[](double x)->double{return 1.0;};
	map(h_b, h_b, n, f);
	double *d_b;
	hipMalloc((void**)&d_b, n*sizeof(double));
	hipMemcpy(d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);

	alpha=-ua;
	hipblasDaxpy(cublasH, n, &alpha, d_D2+n*(n-1), 1, d_b, 1);
	alpha=-ub;
	hipblasDaxpy(cublasH, n, &alpha, d_D2        , 1, d_b, 1);
	hipMemcpy(d_b,     &ub, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b+n-1, &ua, sizeof(double), hipMemcpyHostToDevice);

	// solve Poisson D2*u=f(x)
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);
	int lwork;
	int m=n-2, lda=n, ldb=n, nrhs=1;

	hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_D2+n+1, lda, &lwork);

	int    *d_info; hipMalloc((void**)&d_info, sizeof(int));
	int    *d_Ipiv; hipMalloc((void**)&d_Ipiv, m*sizeof(int));
	double *d_Work; hipMalloc((void**)&d_Work, lwork*sizeof(double));

	hipsolverDnDgetrf(cusolverH, m, m, d_D2+n+1, lda, d_Work, d_Ipiv, d_info);
	hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, nrhs, d_D2+n+1, lda, d_Ipiv, d_b+1, ldb, d_info);

	// display u
	hipMemcpy(h_b, d_b, n*sizeof(double), hipMemcpyDeviceToHost);
	disp(h_b, n, 1, 1);
	free(h_b);

	// free auxiliary variables
	hipFree(d_Work);
	hipFree(d_Ipiv);
	hipFree(d_info);

	// free device memory
	hipFree(d_x);
	hipFree(d_D);
	hipFree(d_D2);
	hipFree(d_b);

	// destroy library handles
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);
}

int main(int argc, char **argv){
	poisson(-1, 1, 16);
	return 0;
}
