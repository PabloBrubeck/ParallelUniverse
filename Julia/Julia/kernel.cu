#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <algorithm>

#include "book.h"
#include "gl_helper.h"

#define WIDTH 1024
#define HEIGHT 512
#define PI 3.1415926535898

struct hipComplex {
    double   re, im;
    __device__ hipComplex( double a, double b ) : re(a), im(b)  {}
    __device__ double magnitude2( void ) {
        return re*re+im*im;
    }
    __device__ 
    hipComplex operator*(const hipComplex& z) {
        return hipComplex(re*z.re-im*z.im, im*z.re+re*z.im);
    }
    __device__
    hipComplex operator+(const hipComplex& z) {
        return hipComplex(re+z.re, im+z.im);
    }
};

__device__
int julia(double jx, double jy) {
	hipComplex c(jx, jy);
	hipComplex z(jx, jy);
	int i=0;
	do{
		z=z*z+c;
		i++;
	}while(z.magnitude2()<4 && i<258);
	return i;
}

__global__
void kernel(unsigned char *ptr, const double h, const double k, const double zoom) {
	// map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x>=WIDTH || y>=HEIGHT){
		return;
	}
	int offset=y*WIDTH+x;

	// now calculate the value at that position
	double range=min(HEIGHT,WIDTH)/2;
	double jx=zoom*(x-h-WIDTH/2)/range;
	double jy=zoom*(HEIGHT/2-y-k)/range;
	
	int i=julia(jx, jy);
	
	double r=max(0, 256-4*abs(i-192));
	double g=max(0, 256-4*abs(i-64));
	double b=max(0, 256-4*i);
	ptr[4*offset+0]=(int)(r*r/255);
	ptr[4*offset+1]=(int)(g*g/255);
	ptr[4*offset+2]=(int)(b*b/255);
	ptr[4*offset+3]=255;
}

struct CPUBitmap {

	unsigned char *pixels;
	unsigned char *dev_bitmap;
    int x, y;
	int h, k;
	int htemp, ktemp;
	double zoom;

    void *dataBlock;
    void (*bitmapExit)(void*);

    CPUBitmap( int width, int height) {
		pixels=new unsigned char[4*width*height];
		x=width;
        y=height;
		HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, image_size()));
		h=0;
		k=0;
		zoom=1.0f;
    }

    ~CPUBitmap() {
        delete [] pixels;
		HANDLE_ERROR(hipFree(dev_bitmap));
    }

    unsigned char* get_ptr( void ) const   { 
		return pixels; 
	}
    long image_size( void ) const { 
		return x * y * 4; 
	}

    void display_and_exit( void(*e)(void*) = NULL ) {
        CPUBitmap**   bitmap = get_bitmap_ptr();
        *bitmap = this;
        bitmapExit = e;
        // a bug in the Windows GLUT implementation prevents us from
        // passing zero arguments to glutInit()
        int c=1;
        char* dummy = "";
        glutInit( &c, &dummy );
        glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );
        glutInitWindowSize( x, y );
        glutCreateWindow( "bitmap" );
        glutKeyboardFunc(Key);
		glutMouseFunc(Mouse);
		glutMotionFunc(Motion);
        glutDisplayFunc(Draw);
        glutMainLoop();
    }

     // static method used for glut callbacks
    static CPUBitmap** get_bitmap_ptr( void ) {
        static CPUBitmap *gBitmap;
        return &gBitmap;
    }

    // static methods used for glut callbacks
    static void Key(unsigned char key, int x, int y) {
		CPUBitmap *bitmap = *(get_bitmap_ptr());
		int xm=x-WIDTH/2;
        int ym=y-HEIGHT/2;
		double scale=1.1;
		switch (key) {
			case 'z':
				bitmap->zoom/=scale;
				bitmap->k=(int)((bitmap->k-ym)*scale+ym+0.5);
				bitmap->h=(int)((bitmap->h-xm)*scale+xm+0.5);
				Draw();
				break;
		    case 'x':
				bitmap->zoom*=scale;
				bitmap->k=(int)((bitmap->k-ym)/scale+ym+0.5);
				bitmap->h=(int)((bitmap->h-xm)/scale+xm+0.5);
				Draw();
				break;
            case 27:
                if (bitmap->dataBlock != NULL && bitmap->bitmapExit != NULL)
                    bitmap->bitmapExit( bitmap->dataBlock );
                exit(0);
        }
    }
	static void Mouse(int button, int state, int x, int y){
		if(state==GLUT_DOWN){
			CPUBitmap* bitmap=*(get_bitmap_ptr());
			bitmap->htemp=x;
			bitmap->ktemp=y;
		}
	}
	static void Motion(int x, int y){
		CPUBitmap* bitmap=*(get_bitmap_ptr());

		bitmap->h+=x-bitmap->htemp;
		bitmap->k+=y-bitmap->ktemp;

		bitmap->htemp=x;
		bitmap->ktemp=y;

		Draw();
	}

    // static method used for glut callbacks
    static void Draw(void) {
        CPUBitmap* bitmap=*(get_bitmap_ptr());
		size_t size=bitmap->image_size();

		dim3 blockSize(32, 16);
		dim3 gridSize((WIDTH+blockSize.x-1)/blockSize.x, (HEIGHT+blockSize.y-1)/blockSize.y);
		kernel<<<gridSize, blockSize>>>(bitmap->dev_bitmap, bitmap->h, bitmap->k, bitmap->zoom);
		HANDLE_ERROR(hipMemcpy(bitmap->pixels, bitmap->dev_bitmap, size, hipMemcpyDeviceToHost));
        glDrawPixels(bitmap->x, bitmap->y, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
        glFlush();
    }
};

int main( void ) {
    CPUBitmap bitmap(WIDTH, HEIGHT);                              
    bitmap.display_and_exit();
}