#include "hip/hip_runtime.h"
#include "multShare.h"

using namespace std;

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// Load A and B to device memory
	Matrix d_A;
	d_A.width=A.width;
	d_A.height=A.height;
	size_t size=A.width*A.height*sizeof(float);
	hipError_t err=hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	err=hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));
	
	// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));
	
	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((B.width+dimBlock.x-1)/dimBlock.x, (A.height+dimBlock.y-1)/dimBlock.y);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));
	
	// Read C from device memory
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(err));
	
	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	// hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > A.height || col > B.width){
		return;
	}
	for(int i = 0; i<A.width; i++){
		Cvalue+=A.elements[row*A.width+i]*B.elements[i*B.width+col];
	}
	C.elements[row*C.width+col] = Cvalue;
}
// Usage: multNoShare a1 a2 b2

void printMatrix(Matrix M, int n){
	for (int i=0; i<min(n, M.height); i++){
		for (int j = 0; j < min(n, M.width); j++){
			printf("%f ", M.elements[i*M.width + j]);
		}
		printf("\n");
	}
}

int main(){
	Matrix A, B, C;
	int a1, a2, b1, b2;
	// Read some values from the commandline
	a1 = 5; /* Height of A */
	a2 = 3 ; /* Width of A */
	b1 = a2; /* Height of B */
	b2 = 2; /* Width of B */
	A.height=a1;
	A.width=a2;
	A.elements=new float[A.width*A.height*sizeof(float)];
	B.height=b1;
	B.width=b2;
	B.elements=new float[B.width*B.height*sizeof(float)];
	C.height=A.height;
	C.width=B.width;
	C.elements=new float[C.width*C.height*sizeof(float)];
	for(int i=0; i<A.height; i++)
		for(int j=0; j<A.width; j++)
			A.elements[i*A.width+j]=(float)(rand()%3);
	for(int i = 0; i < B.height; i++)
		for (int j = 0; j < B.width; j++)
			B.elements[i*B.width+j]=(float)(rand()%2);
	MatMul(A, B, C);
	// Print up to a 10x10 portion of the three matrices
	printMatrix(A, 10);
	printf("\n");
	printMatrix(B, 10);
	printf("\n");
	printMatrix(C, 10);
	printf("\n");
}