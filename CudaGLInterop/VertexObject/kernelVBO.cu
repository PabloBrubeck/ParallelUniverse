#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <cutil_math.h>
#include "geometry.cuh"
#include "hand.cuh"

#define MAXTHREADS 512

__global__
void animate(float4 *d_vertex, float4 *d_shape, dim3 mesh){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	int k=blockIdx.z*blockDim.z+threadIdx.z;
	if(i<mesh.x && j<mesh.y && k<mesh.z){
		int gid=(k*mesh.y+j)*mesh.x+i;
		d_vertex[gid]=0.99f*d_vertex[gid]+0.01f*d_shape[gid];
	}
}

int ceil(int num, int den){
	return (num+den-1)/den;
}

void printArray(float4* arr, int n){
	for(int i=0; i<n; i++){
		printf("%f\t %f\t %f\t %f\n", arr[i].x, arr[i].y, arr[i].z, arr[i].w);
	}
}

void launch_kernel(float4 *d_pos, float4 *d_norm, uchar4 *d_color, dim3 mesh, float time){
	static const int n=mesh.x*mesh.y*mesh.z;
	
	
	if(time==0.f){
		float4* h=new float4[25];
		position(h);
		printArray(h, 25);
		hipMemcpy(d_pos, h, 25*sizeof(float4), hipMemcpyHostToDevice);
		hipMemset(d_color, 255u, 25*sizeof(unsigned int));
	}
}